
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <string>
#include <algorithm>
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <dirent.h>

using namespace std;

void writeFromUChar(unsigned char, unsigned char &, int, FILE *);
long int sizeOfTheFile(char *);
void writeFileSize(long int, unsigned char &, int, FILE *);
void writeFileContent(FILE *, long int, string *, unsigned char &, int &, FILE *);

struct TreeNode
{ // this structure will be used to create the translation tree
    TreeNode *left, *right;
    long int occurrences;
    unsigned char character;
    string bit;
};

bool TreeNodeCompare(TreeNode a, TreeNode b)
{
    return a.occurrences < b.occurrences;
}

int main(int argc, char *argv[])
{
    long int freqCount[256] = {0};
    int uniqueSymbolCount = 0;
    if (argc != 2)
    {
        std::cout << "Must provide a single file name." << endl;
        return 0;
    }

    FILE *originalFilePtr;
    originalFilePtr = fopen(argv[1], "rb");
    if (!originalFilePtr)
    {
        std::cout << argv[1] << " file does not exist" << endl
                  << "Process has been terminated" << endl;
        return 0;
    }
    fclose(originalFilePtr);

    // Histograming the frequency of bytes.
    unsigned char *readBufPtr, readBuf;
    readBufPtr = &readBuf;

    long int originalFileSize = sizeOfTheFile(argv[1]);
    std::cout << "The size of the sum of ORIGINAL files is: " << originalFileSize << " bytes" << endl;

    // "rb" is for reading binary files
    originalFilePtr = fopen(argv[1], "rb");
    // reading the first byte of the file into readBuf.
    fread(readBufPtr, 1, 1, originalFilePtr);
    for (long int i = 0; i < originalFileSize; i++)
    { // counting usage frequency of unique bytes inside the file
        freqCount[readBuf]++;
        fread(readBufPtr, 1, 1, originalFilePtr);
    }
    fclose(originalFilePtr);

    // Traverse through all possible bytes and count the number of unique bytes.
    for (long int *i = freqCount; i < freqCount + 256; i++)
    {
        if (*i)
        {
            uniqueSymbolCount++;
        }
    }

    // Step 1: Initialize the leaf nodes for Huffman tree construction.
    // Each leaf node represents a unique byte and its frequency in the input data.
    TreeNode nodesForHuffmanTree[uniqueSymbolCount * 2 - 1];
    TreeNode *currentNode = nodesForHuffmanTree;

    // Step 2: Fill the array with data for each unique byte.
    for (long int *frequency = freqCount; frequency < freqCount + 256; frequency++)
    {
        if (*frequency)
        {
            currentNode->right = NULL;
            currentNode->left = NULL;
            currentNode->occurrences = *frequency;
            currentNode->character = frequency - freqCount;
            currentNode++;
        }
    }

    // Step 3: Sort the leaf nodes based on frequency to prepare for tree construction.
    // In ascending order.
    sort(nodesForHuffmanTree, nodesForHuffmanTree + uniqueSymbolCount, TreeNodeCompare);

    // Step 4: Construct the Huffman tree by merging nodes with the lowest frequencies.
    TreeNode *smallestNode = nodesForHuffmanTree;
    TreeNode *secondSmallestNode = nodesForHuffmanTree + 1;
    TreeNode *newInternalNode = nodesForHuffmanTree + uniqueSymbolCount;
    TreeNode *nextInternalNode = nodesForHuffmanTree + uniqueSymbolCount;
    TreeNode *nextLeafNode = nodesForHuffmanTree + 2;
    for (int i = 0; i < uniqueSymbolCount - 1; i++)
    {
        // Create a new internal node that combines the two smallest nodes.
        newInternalNode->occurrences = smallestNode->occurrences + secondSmallestNode->occurrences;
        newInternalNode->left = smallestNode;
        newInternalNode->right = secondSmallestNode;
        // Assign bits for tree navigation: '1' for the path to smallestNode,
        // '0' for secondSmallestNode.
        smallestNode->bit = "1";
        secondSmallestNode->bit = "0";
        newInternalNode++;

        // Update smallestNode and secondSmallestNode for the next iteration.
        if (nextLeafNode >= nodesForHuffmanTree + uniqueSymbolCount)
        {
            // All leaf nodes have been processed; proceed with internal nodes.
            smallestNode = nextInternalNode;
            nextInternalNode++;
        }
        else
        {
            // Choose the next smallest node from the leaf or internal nodes.
            smallestNode = (nextLeafNode->occurrences < nextInternalNode->occurrences) ? nextLeafNode++ : nextInternalNode++;
        }

        // Repeat the process for secondSmallestNode.
        if (nextLeafNode >= nodesForHuffmanTree + uniqueSymbolCount)
        {
            secondSmallestNode = nextInternalNode;
            nextInternalNode++;
        }
        else if (nextInternalNode >= newInternalNode)
        {
            secondSmallestNode = nextLeafNode;
            nextLeafNode++;
        }
        else
        {
            secondSmallestNode = (nextLeafNode->occurrences < nextInternalNode->occurrences) ? nextLeafNode++ : nextInternalNode++;
        }
    }

    // Step 5: Assign Huffman codes to each node.
    // Iterate from the last internal node to the root, building the Huffman codes in reverse.
    for (TreeNode *node = nodesForHuffmanTree + uniqueSymbolCount * 2 - 2; node > nodesForHuffmanTree - 1; node--)
    {
        // If a left child exists, concatenate the current node's code to it. This assigns the '0' path.
        if (node->left)
        {
            node->left->bit = node->bit + node->left->bit;
        }

        // Similar operation for the right child, representing the '1' path.
        if (node->right)
        {
            node->right->bit = node->bit + node->right->bit;
        }
    }

    string scompressed = argv[1];
    scompressed += ".compressed";
    FILE *compressedFilePtr = fopen(&scompressed[0], "wb");

    // Writing the first piece of header information: the count of unique bytes.
    // This count is essential for reconstructing the Huffman tree during the decompression process.
    fwrite(&uniqueSymbolCount, 1, 1, compressedFilePtr);

    int bitCounter = 0;
    unsigned char bufferByte;
    // Initializing a pointer for iterating through the transformation strings.
    char *transformationStringPtr;
    // Variables for storing the length of the transformation string and the current character being processed.
    unsigned char transformationLength, currentCharacter;
    // Array to store transformation strings for each unique character to optimize compression.
    string transformationStrings[256];

    // Iterate through each node in the Huffman tree to write transformation codes to the compressed file.
    for (TreeNode *node = nodesForHuffmanTree; node < nodesForHuffmanTree + uniqueSymbolCount; node++)
    {
        // Store the transformation string for the current character in the array.
        transformationStrings[node->character] = node->bit;
        transformationLength = node->bit.length();
        currentCharacter = node->character;

        // Write the current character and its transformation string length to the compressed file.
        writeFromUChar(currentCharacter, bufferByte, bitCounter, compressedFilePtr);
        writeFromUChar(transformationLength, bufferByte, bitCounter, compressedFilePtr);

        // Write the transformation string bit by bit to the compressed file.
        transformationStringPtr = &node->bit[0];
        while (*transformationStringPtr)
        {
            bufferByte <<= 1;
            if (*transformationStringPtr == '1')
            {
                bufferByte |= 1;
            }
            bitCounter++;
            transformationStringPtr++;
            if (bitCounter == 8)
            {
                fwrite(&bufferByte, 1, 1, compressedFilePtr);
                bitCounter = 0;
            }
        }
    }

    originalFilePtr = fopen(argv[1], "rb");

    // Writing the size of the file, its name, and its content in the compressed format.
    writeFileSize(originalFileSize, bufferByte, bitCounter, compressedFilePtr);
    writeFileContent(originalFilePtr, originalFileSize, transformationStrings, bufferByte, bitCounter, compressedFilePtr);
    fclose(originalFilePtr);

    // Ensuring the last byte is written to the compressed file by aligning the bit counter.
    if (bitCounter > 0)
    {
        bufferByte <<= (8 - bitCounter);
        fwrite(&bufferByte, 1, 1, compressedFilePtr);
    }

    fclose(compressedFilePtr);

    // Get the size of compressed file.
    long int compressedFileSize = sizeOfTheFile(&scompressed[0]);
    std::cout << "The size of the COMPRESSED file is: " << compressedFileSize << " bytes" << endl;

    // Calculate the compression ratio.
    float compressionRatio = 100.0f * static_cast<float>(compressedFileSize) / static_cast<float>(originalFileSize);
    std::cout << "Compressed file's size is [" << compressionRatio << "%] of the original files." << endl;

    // Warning if the compressed file is unexpectedly larger than the original sum.
    if (compressedFileSize > originalFileSize)
    {
        std::cout << "\nWARNING: The compressed file's size is larger than the sum of the originals.\n\n";
    }

    std::cout << endl
              << "Created compressed file: " << scompressed << endl;
    std::cout << "Compression is complete" << endl;
}

// below function is used for writing the uChar to compressed file
// It does not write it directly as one byte instead it mixes uChar and current byte, writes 8 bits of it
// and puts the rest to curent byte for later use
void writeFromUChar(unsigned char byteToWrite, unsigned char &bufferByte, int bitCounter, FILE *filePtr)
{
    // Going to write at least 1 byte, first shift the bufferByte to the left
    // to make room for the new byte.
    bufferByte <<= 8 - bitCounter;
    bufferByte |= (byteToWrite >> bitCounter);
    fwrite(&bufferByte, 1, 1, filePtr);
    bufferByte = byteToWrite;
}

// This function is writing byte count of current input file to compressed file using 8 bytes
// It is done like this to make sure that it can work on little, big or middle-endian systems
void writeFileSize(long int fileSize, unsigned char &bufferByte, int bitCounter, FILE *filePtr)
{
    for (int i = 0; i < 8; i++)
    {
        writeFromUChar(fileSize % 256, bufferByte, bitCounter, filePtr);
        fileSize /= 256;
    }
}

// Below function translates and writes bytes from current input file to the compressed file.
void writeFileContent(FILE *originalFilePtr, long int originalFileSize, string *transformationStrings, unsigned char &bufferByte, int &bitCounter, FILE *compressedFilePtr)
{
    unsigned char *bufPtr, buf;
    bufPtr = &buf;
    char *strPointer;
    fread(bufPtr, 1, 1, originalFilePtr);
    for (long int i = 0; i < originalFileSize; i++)
    {
        strPointer = &transformationStrings[buf][0];
        while (*strPointer)
        {
            if (bitCounter == 8)
            {
                fwrite(&bufferByte, 1, 1, compressedFilePtr);
                bitCounter = 0;
            }
            switch (*strPointer)
            {
            case '1':
                bufferByte <<= 1;
                bufferByte |= 1;
                bitCounter++;
                break;
            case '0':
                bufferByte <<= 1;
                bitCounter++;
                break;
            default:
                cout << "An error has occurred" << endl
                     << "Process has been aborted";
                exit(2);
            }
            strPointer++;
        }
        fread(bufPtr, 1, 1, originalFilePtr);
    }
}

long int sizeOfTheFile(char *path)
{
    ifstream file(path, ifstream::ate | ifstream::binary);
    return file.tellg();
}